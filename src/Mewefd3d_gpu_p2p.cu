#include "hip/hip_runtime.h"
// TODO:
// - make wave field arrays global on CPU
// - one FDM3D structure describing the old wave fields, also make it a global variable
// - at the begining of each time block, interpolate the wave fields to meet the requirement of current grid size
// - copy the wave fields on CPU to GPU according to nyinterior
// - after interpolation, the old FDM3D points to the current FDM3D
// - at the last time step of each time block main loop, update the global wavefield arrays

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <rsf.h>
#include "fdutil.h"
#include "vel.h"
#include "box.h"
#include "resample.h"
}

#include "ewefd3d_kernels.h"

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define NOP 4 /* derivative operator half-size */


fdm3d clonefdm(const fdm3d &fdm)
{
  sf_axis az = sf_maxa(fdm->nz, fdm->oz, fdm->dz);
  sf_axis ax = sf_maxa(fdm->nx, fdm->ox, fdm->dx);
  sf_axis ay = sf_maxa(fdm->ny, fdm->oy, fdm->dy);

  return fdutil3d_init(fdm->verb,fdm->free,az,ax,ay,fdm->nb,fdm->ompchunk);
}

static void init_host_umo(const fdm3d &fdm, float ***&h_umx, float ***&h_uox,  float ***&h_umy,  float ***&h_uoy,  float ***&h_umz,  float ***&h_uoz)
{
  int n1 = fdm->nzpad; int n2 = fdm->nxpad; int n3 = fdm->nypad;
  int bytes = n1 * n2 * n3 * sizeof(float);

  h_umz = sf_floatalloc3(n1, n2, n3); memset(h_umz[0][0], 0, bytes);
  h_umx = sf_floatalloc3(n1, n2, n3); memset(h_umx[0][0], 0, bytes);
  h_umy = sf_floatalloc3(n1, n2, n3); memset(h_umy[0][0], 0, bytes);
  h_uoz = sf_floatalloc3(n1, n2, n3); memset(h_uoz[0][0], 0, bytes);
  h_uox = sf_floatalloc3(n1, n2, n3); memset(h_uox[0][0], 0, bytes);
  h_uoy = sf_floatalloc3(n1, n2, n3); memset(h_uoy[0][0], 0, bytes);
}

static void release_host_umo(float ***&h_umx, float ***&h_uox,  float ***&h_umy,  float ***&h_uoy,  float ***&h_umz,  float ***&h_uoz)
{
  free(**h_umx); free(*h_umx); free(h_umx);
  free(**h_umz); free(*h_umz); free(h_umz);
  free(**h_umy); free(*h_umy); free(h_umy);
  free(**h_uox); free(*h_uox); free(h_uox);
  free(**h_uoz); free(*h_uoz); free(h_uoz);
  free(**h_uoy); free(*h_uoy); free(h_uoy);
}


// checks the current GPU device for an error flag and prints to stderr
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
     if (hipSuccess != err)
        sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

static void update_axis(const fdm3d &fdm, sf_axis &az, sf_axis &ax, sf_axis &ay, bool verb) {
  sf_setn(az,fdm->nzpad); sf_seto(az,fdm->ozpad); if(verb) sf_raxa(az);
  sf_setn(ax,fdm->nxpad); sf_seto(ax,fdm->oxpad); if(verb) sf_raxa(ax);
  sf_setn(ay,fdm->nypad); sf_seto(ay,fdm->oypad); if(verb) sf_raxa(ay);
}

static float **setup_bell(int nbell, int ngpu)
{
  /*------------------------------------------------------------*/
  /* setup bell for source injection smoothing */
  if (nbell * 2 + 1 > 32){
    sf_error("nbell must be <= 15\n");
  }

  float *h_bell;
  h_bell = (float*)malloc((2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float));

  float s = 0.5*nbell;
  for (int iy=-nbell;iy<=nbell;iy++) {
    for (int ix=-nbell;ix<=nbell;ix++) {
      for(int iz=-nbell;iz<=nbell;iz++) {
        h_bell[(iy + nbell) * (2*nbell+1) * (2*nbell+1) + (iz + nbell) * (2*nbell+1) + (ix + nbell)] = exp(-(iz*iz+ix*ix+iy*iy)/s);
      }
    }
  }

  // copy bell coeficients to the GPUs
  float **d_bell = (float**)malloc(ngpu*sizeof(float*));
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_bell[g], (2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float));
    sf_check_gpu_error("hipMalloc d_bell");
    hipMemcpy(d_bell[g], h_bell, (2*nbell+1)*(2*nbell+1)*(2*nbell+1)*sizeof(float), hipMemcpyDefault);
    sf_check_gpu_error("copy d_bell to device");
  }

  free(h_bell);

  return d_bell;
  /*------------------------------------------------------------*/
}
static void setup_output_data(sf_file &Fdat, sf_axis &at, const sf_axis &ar, const sf_axis &ac, int nt, int jdata, float dt)
{
  /*------------------------------------------------------------*/
  /* setup output data files and arrays */
  sf_oaxa(Fdat,ar,1);
  sf_oaxa(Fdat,ac,2);

  sf_setn(at,nt/jdata);
  sf_setd(at,dt*jdata);
  sf_oaxa(Fdat,at,3);
}

static void set_output_wfd(sf_file &Fwfl, sf_axis &at, const sf_axis &az, const sf_axis &ax, const sf_axis &ay, const sf_axis &ac, int nt, float dt, int jsnap, bool verb)
{
  int ntsnap=0;
  for(int it=0; it<nt; it++) {
    if(it%jsnap==0) ntsnap++;
  }
  sf_setn(at,  ntsnap);
  sf_setd(at,dt*jsnap);
  if(verb) sf_raxa(at);

  sf_oaxa(Fwfl,az,1);
  sf_oaxa(Fwfl,ax,2);
  sf_oaxa(Fwfl,ay,3);
  sf_oaxa(Fwfl,ac, 4);
  sf_oaxa(Fwfl,at, 5);
}
static void alloc_wlf(const fdm3d &fdm, float *** &uoz, float *** &uox, float *** &uoy, float *&h_uoz, float *&h_uox, float *&h_uoy, float ***&uc, int nyinterior)
{
    // Used to accumulate wavefield data from other GPUs
    uoz=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
    uox=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
    uoy=sf_floatalloc3(fdm->nzpad,fdm->nxpad,fdm->nypad);
    h_uoz = (float*)malloc(nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    h_uox = (float*)malloc(nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    h_uoy = (float*)malloc(nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));

    //uc=sf_floatalloc3(sf_n(az),sf_n(ax),sf_n(ay));
    uc=sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);

}
static float **init_wavelet(sf_file &Fwav, int ns, int nc, int nt, int ngpu)
{
  /*------------------------------------------------------------*/
  /* read source wavelet(s) and copy to each GPU (into d_ww) */
  float ***ww=sf_floatalloc3(ns,nc,nt);
  sf_floatread(ww[0][0],nt*nc*ns,Fwav);

  float *h_ww = (float*)malloc(ns*nc*nt*sizeof(float));
  for (int t = 0; t < nt; t++){
    for (int c = 0; c < nc; c++){
      for (int s = 0; s < ns; s++){
        h_ww[t * nc * ns + c * ns + s]=ww[t][c][s];
      }
    }
  }

  float **d_ww = (float**)malloc(ngpu*sizeof(float*));
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_ww[g], ns*nc*nt*sizeof(float));
    sf_check_gpu_error("hipMalloc source wavelet to device");
    hipMemcpy(d_ww[g], h_ww, ns*nc*nt*sizeof(float), hipMemcpyDefault);
    sf_check_gpu_error("copy source wavelet to device");
  }

  free(**ww); free(*ww); free(ww);
  return d_ww;
  /*------------------------------------------------------------*/
}
static void setup_output_array(float *&h_dd, float *&h_dd_combined, float **&d_dd, int ngpu, int nr, int nc)
{
  /*------------------------------------------------------------*/
  /* data array */
  h_dd = (float*)malloc(nr * nc * sizeof(float));
  h_dd_combined = (float*)malloc(nr * nc * sizeof(float));

  d_dd = (float**)malloc(ngpu*sizeof(float*));
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_dd[g], nr*nc*sizeof(float));
    sf_check_gpu_error("allocate data arrays");
  }
  /*------------------------------------------------------------*/

}
static void setup_src_rcv_cord(sf_file &Fsou, sf_file &Frec, pt3d *&ss, pt3d *&rr, int ns, int nr)
{
  /*------------------------------------------------------------*/
  /* setup source/receiver coordinates */
  ss = (pt3d*) sf_alloc(ns,sizeof(*ss));
  rr = (pt3d*) sf_alloc(nr,sizeof(*rr));

  pt3dread1(Fsou,ss,ns,3); /* read (x,y,z) coordinates */
  pt3dread1(Frec,rr,nr,3); /* read (x,y,z) coordinates */

}
static void setup_interp_cooef(float **&d_Sw000, float **&d_Sw001, float **&d_Sw010, float **&d_Sw011, float **&d_Sw100, float **&d_Sw101, float **&d_Sw110, float **&d_Sw111, int **&d_Sjz, int **&d_Sjx, int **&d_Sjy, const fdm3d &fdm, pt3d *ss, int ns, int ngpu)
{
  /* calculate 3d linear interpolation coefficients for source locations and copy to each GPU*/
  lint3d cs = lint3d_make(ns,ss,fdm);
  d_Sw000 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw001 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw010 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw011 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw100 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw101 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw110 = (float**)malloc(ngpu*sizeof(float*));
  d_Sw111 = (float**)malloc(ngpu*sizeof(float*));

  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_Sw000[g], ns * sizeof(float));
    hipMalloc(&d_Sw001[g], ns * sizeof(float));
    hipMalloc(&d_Sw010[g], ns * sizeof(float));
    hipMalloc(&d_Sw011[g], ns * sizeof(float));
    hipMalloc(&d_Sw100[g], ns * sizeof(float));
    hipMalloc(&d_Sw101[g], ns * sizeof(float));
    hipMalloc(&d_Sw110[g], ns * sizeof(float));
    hipMalloc(&d_Sw111[g], ns * sizeof(float));
    sf_check_gpu_error("hipMalloc source interpolation coeficients to device");
    hipMemcpy(d_Sw000[g], cs->w000, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw001[g], cs->w001, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw010[g], cs->w010, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw011[g], cs->w011, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw100[g], cs->w100, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw101[g], cs->w101, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw110[g], cs->w110, ns * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_Sw111[g], cs->w111, ns * sizeof(float), hipMemcpyDefault);
    sf_check_gpu_error("copy source interpolation coeficients to device");

  }

  // z, x, and y coordinates of each source
  d_Sjz = (int**)malloc(ngpu*sizeof(int*));
  d_Sjx = (int**)malloc(ngpu*sizeof(int*));
  d_Sjy = (int**)malloc(ngpu*sizeof(int*));
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_Sjz[g], ns * sizeof(int));
    hipMalloc(&d_Sjx[g], ns * sizeof(int));
    hipMalloc(&d_Sjy[g], ns * sizeof(int));
    sf_check_gpu_error("hipMalloc source coords to device");
    hipMemcpy(d_Sjz[g], cs->jz, ns * sizeof(int), hipMemcpyDefault);
    hipMemcpy(d_Sjx[g], cs->jx, ns * sizeof(int), hipMemcpyDefault);
    hipMemcpy(d_Sjy[g], cs->jy, ns * sizeof(int), hipMemcpyDefault);
    sf_check_gpu_error("copy source coords to device");
  }

}
static void setup_fd_cooef(const fdm3d &fdm, float &idz, float &idx, float &idy)
{
  /*------------------------------------------------------------*/
  /* setup FD coefficients */
  idz = 1/fdm->dz;;
  idx = 1/fdm->dx;
  idy = 1/fdm->dy;
  /*------------------------------------------------------------*/
}

static void read_density_velocity(sf_file &Fden, sf_file &Fccc, const fdm3d &fdm, float ***&h_ro, float ***&h_c11, float ***&h_c22, float ***&h_c33, float ***&h_c44, float ***&h_c55, float ***&h_c66, float ***&h_c12, float ***&h_c13, float ***&h_c23, int nz, int nx, int ny)
{
  /*------------------------------------------------------------*/
  /* read in model density and stiffness arrays */
  float ***tt1 = sf_floatalloc3(nz, nx, ny);

  /* input density */
  h_ro = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fden);     expand_cpu(tt1[0][0], h_ro[0][0], fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);

  /* input stiffness */
  h_c11 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c22 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c33 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c44 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c55 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c66 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c12 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c13 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  h_c23 = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c11[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c22[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c33[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c44[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c55[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c66[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c12[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c13[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  sf_floatread(tt1[0][0],nz*nx*ny,Fccc);    expand_cpu(tt1[0][0],h_c23[0][0],fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
  free(**tt1); free(*tt1); free(tt1);
}

static void copy_den_vel_to_dev(const fdm3d &fdm, float **&d_ro , float **&d_c11, float **&d_c22, float **&d_c33, float **&d_c44, float **&d_c55, float **&d_c66, float **&d_c12, float **&d_c13, float **&d_c23, const float *h_ro, const float *h_c11, const float *h_c22, const float *h_c33, const float *h_c44, const float *h_c55, const float *h_c66, const float *h_c12, const float *h_c13, const float *h_c23, int nyinterior, int ngpu)
{
  d_ro = (float**)malloc(ngpu*sizeof(float*));
  d_c11 = (float**)malloc(ngpu*sizeof(float*));
  d_c22 = (float**)malloc(ngpu*sizeof(float*));
  d_c33 = (float**)malloc(ngpu*sizeof(float*));
  d_c44 = (float**)malloc(ngpu*sizeof(float*));
  d_c55 = (float**)malloc(ngpu*sizeof(float*));
  d_c66 = (float**)malloc(ngpu*sizeof(float*));
  d_c12 = (float**)malloc(ngpu*sizeof(float*));
  d_c13 = (float**)malloc(ngpu*sizeof(float*));
  d_c23 = (float**)malloc(ngpu*sizeof(float*));

  // allocate density and stiffness sub-domain arrays on each GPU and copy the data
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipMalloc(&d_ro[g] , nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c11[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c22[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c33[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c44[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c55[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c66[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c12[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c13[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_c23[g], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float));
    sf_check_gpu_error("hipMalloc density and stiffness to device");

    hipMemcpy(d_ro[g] , h_ro  + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c11[g], h_c11 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c22[g], h_c22 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c33[g], h_c33 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c44[g], h_c44 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c55[g], h_c55 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c66[g], h_c66 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c12[g], h_c12 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c13[g], h_c13 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_c23[g], h_c23 + g * nyinterior * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
    sf_check_gpu_error("copy density and stiffness to device");

  }

}

static void setup_boundary(const fdm3d fdm, float **&d_bzl_s, float **&d_bzh_s, float **&d_bxl_s, float **&d_bxh_s, float **&d_byl_s, float **&d_byh_s, const float *h_ro, const float *h_c55, float &spo, int nyinterior, int ngpu, float dt, bool dabc)
{
  /*------------------------------------------------------------*/
  /* Boundary condition setup */
  d_bzl_s = (float**)malloc(ngpu*sizeof(float*));
  d_bzh_s = (float**)malloc(ngpu*sizeof(float*));
  d_bxl_s = (float**)malloc(ngpu*sizeof(float*));
  d_bxh_s = (float**)malloc(ngpu*sizeof(float*));
  d_byl_s = (float**)malloc(ngpu*sizeof(float*));
  d_byh_s = (float**)malloc(ngpu*sizeof(float*));

  float *h_bzl_s, *h_bzh_s;
  float *h_bxl_s, *h_bxh_s;
  float *h_byl_s, *h_byh_s;

  spo = 0;
  int nb = fdm->nb;
  if(dabc) {

    /* one-way abc setup   */
    float d;
    float *vs1 = (float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));

    for (int iy = 0; iy < fdm->nypad; iy++) {
      for (int ix = 0; ix < fdm->nxpad; ix++) {
        for (int iz = 0; iz < fdm->nzpad; iz++) {
          vs1[iy * fdm->nzpad * fdm->nxpad + iz * fdm->nxpad + ix] = sqrt(h_c55[iy * fdm->nxpad * fdm->nzpad + iz * fdm->nxpad + ix] / h_ro[iy * fdm->nxpad * fdm->nzpad + iz * fdm->nxpad + ix]);
        }
      }
    }

    h_bzl_s = (float*)malloc(fdm->nxpad * nyinterior * sizeof(float));
    h_bzh_s = (float*)malloc(fdm->nxpad * nyinterior * sizeof(float));
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      for (int ix = 0; ix < fdm->nxpad; ix++){
        for (int iy = 0; iy < nyinterior; iy++){
          d = vs1[(g * nyinterior + iy) * fdm->nzpad * fdm->nxpad + NOP * fdm->nxpad + ix] * dt/fdm->dz;
          h_bzl_s[iy * fdm->nxpad + ix] = (1-d)/(1+d);
          d = vs1[(g * nyinterior + iy) * fdm->nzpad * fdm->nxpad + (fdm->nzpad-NOP-1) * fdm->nxpad + ix] * dt/fdm->dz;
          h_bzh_s[iy * fdm->nxpad + ix] = (1-d)/(1+d);
        }
      }

      hipMalloc(&d_bzl_s[g], fdm->nxpad * nyinterior * sizeof(float));
      hipMalloc(&d_bzh_s[g], fdm->nxpad * nyinterior * sizeof(float));
      hipMemcpy(d_bzl_s[g], h_bzl_s, fdm->nxpad * nyinterior * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_bzh_s[g], h_bzh_s, fdm->nxpad * nyinterior * sizeof(float), hipMemcpyDefault);
    }


    h_bxl_s = (float*)malloc(fdm->nzpad * nyinterior * sizeof(float));
    h_bxh_s = (float*)malloc(fdm->nzpad * nyinterior * sizeof(float));
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      for (int iz = 0; iz < fdm->nzpad; iz++){
        for (int iy = 0; iy < nyinterior; iy++){
          d = vs1[(g * nyinterior + iy) * fdm->nzpad * fdm->nxpad + iz * fdm->nxpad + NOP] *dt/fdm->dx;
          h_bxl_s[iy * fdm->nzpad + iz] = (1-d)/(1+d);
          d = vs1[(g * nyinterior + iy) * fdm->nzpad * fdm->nxpad + iz * fdm->nxpad + (fdm->nxpad-NOP-1)] *dt/fdm->dx;
          h_bxh_s[iy * fdm->nzpad + iz] = (1-d)/(1+d);
        }
      }
      hipMalloc(&d_bxl_s[g], fdm->nzpad * nyinterior * sizeof(float));
      hipMalloc(&d_bxh_s[g], fdm->nzpad * nyinterior * sizeof(float));
      hipMemcpy(d_bxl_s[g], h_bxl_s, fdm->nzpad * nyinterior * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_bxh_s[g], h_bxh_s, fdm->nzpad * nyinterior * sizeof(float), hipMemcpyDefault);
    }


    h_byl_s = (float*)malloc(fdm->nzpad * fdm->nxpad * sizeof(float));
    h_byh_s = (float*)malloc(fdm->nzpad * fdm->nxpad * sizeof(float));
    for (int ix = 0; ix < fdm->nxpad; ix++){
      for (int iz = 0; iz < fdm->nzpad; iz++){
        d = vs1[NOP * fdm->nzpad * fdm->nxpad + iz * fdm->nxpad + ix] *dt/fdm->dy;
        h_byl_s[ix * fdm->nzpad + iz] = (1-d)/(1+d);
        d = vs1[(fdm->nypad-NOP-1) * fdm->nzpad * fdm->nxpad + iz * fdm->nxpad + ix] *dt/fdm->dy;
        h_byh_s[ix * fdm->nzpad + iz] = (1-d)/(1+d);
      }
    }
    hipSetDevice(0);
    hipMalloc(&d_byl_s[0], fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemcpy(d_byl_s[0], h_byl_s, fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);

    hipSetDevice(ngpu-1);
    hipMalloc(&d_byh_s[ngpu-1], fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemcpy(d_byh_s[ngpu-1], h_byh_s, fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);

    sf_check_gpu_error("set up ABC coefficients");

    /* sponge set up */
    // sponge coefficients are calculated inside the sponge kernel on GPU based on spo
    spo = (sqrt(2.0) * 4.0f * nb);

    free(h_bzl_s); free(h_bzh_s);
    free(h_bxl_s); free(h_bxh_s);
    free(h_byl_s); free(h_byh_s);
    free(vs1);
  }
  /*------------------------------------------------------------*/
}

static void init_wfd_array(const fdm3d &fdm, float **&d_umx, float **&d_uox, float **&d_upx, float **&d_uax, float **&d_utx, float **&d_umy, float **&d_uoy, float **&d_upy, float **&d_uay, float **&d_uty, float **&d_umz, float **&d_uoz, float **&d_upz, float **&d_uaz, float **&d_utz, float **&d_tzz, float **&d_txx, float **&d_tyy, float **&d_txy, float **&d_tyz, float **&d_tzx, const int *nylocal, int ngpu)
{
  /*------------------------------------------------------------*/
  /* displacement: um = U @ t-1; uo = U @ t; up = U @ t+1 */
  d_umx = (float **)malloc(ngpu*sizeof(float*));
  d_uox = (float **)malloc(ngpu*sizeof(float*));
  d_upx = (float **)malloc(ngpu*sizeof(float*));
  d_uax = (float **)malloc(ngpu*sizeof(float*));
  d_utx = (float **)malloc(ngpu*sizeof(float*));

  d_umy = (float **)malloc(ngpu*sizeof(float*));
  d_uoy = (float **)malloc(ngpu*sizeof(float*));
  d_upy = (float **)malloc(ngpu*sizeof(float*));
  d_uay = (float **)malloc(ngpu*sizeof(float*));
  d_uty = (float **)malloc(ngpu*sizeof(float*));

  d_umz = (float **)malloc(ngpu*sizeof(float*));
  d_uoz = (float **)malloc(ngpu*sizeof(float*));
  d_upz = (float **)malloc(ngpu*sizeof(float*));
  d_uaz = (float **)malloc(ngpu*sizeof(float*));
  d_utz = (float **)malloc(ngpu*sizeof(float*));

  d_tzz = (float **)malloc(ngpu*sizeof(float*));
  d_txx = (float **)malloc(ngpu*sizeof(float*));
  d_tyy = (float **)malloc(ngpu*sizeof(float*));
  d_txy = (float **)malloc(ngpu*sizeof(float*));
  d_tyz = (float **)malloc(ngpu*sizeof(float*));
  d_tzx = (float **)malloc(ngpu*sizeof(float*));

  // allocate and initialize displacement, accel, and stress/strain arrasys to 0 on each GPU
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);

    hipMalloc(&d_umx[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uox[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_upx[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uax[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMalloc(&d_umy[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uoy[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_upy[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uay[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMalloc(&d_umz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uoz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_upz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_uaz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMalloc(&d_tzz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_tyy[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_txx[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_txy[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_tyz[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMalloc(&d_tzx[g], nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    sf_check_gpu_error("allocate grid arrays");


    hipMemset(d_umx[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uox[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_upx[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uax[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMemset(d_umy[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uoy[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_upy[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uay[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMemset(d_umz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uoz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_upz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_uaz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    hipMemset(d_tzz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_tyy[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_txx[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_txy[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_tyz[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));
    hipMemset(d_tzx[g], 0, nylocal[g] * fdm->nzpad * fdm->nxpad * sizeof(float));

    sf_check_gpu_error("initialize grid arrays");
  }
}
static void precompute(const fdm3d &fdm, float **&d_ro, float dt, int nyinterior, int ngpu)
{
  /*------------------------------------------------------------*/
  /* precompute 1/ro * dt^2                     */
  /*------------------------------------------------------------*/
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    dim3 dimGrid1(ceil(fdm->nxpad/8.0f),ceil(fdm->nzpad/8.0f),ceil(nyinterior/8.0f));
    dim3 dimBlock1(8,8,8);
    computeRo<<<dimGrid1, dimBlock1>>>(d_ro[g], dt, fdm->nxpad, fdm->nzpad, nyinterior);
  }
  sf_check_gpu_error("computeRo Kernel");
}

static void main_loop(sf_file Fwfl, sf_file Fdat, const fdm3d fdm, float **d_umx, float **d_uox, float **d_upx, float **d_uax, float **d_utx, float **d_umy, float **d_uoy, float **d_upy, float **d_uay, float **d_uty, float **d_umz, float **d_uoz, float **d_upz, float **d_uaz, float **d_utz, float **d_tzz, float **d_txx, float **d_tyy, float **d_txy, float **d_tyz, float **d_tzx, float **d_c11, float **d_c22, float **d_c33, float **d_c44, float **d_c55, float **d_c66, float **d_c12, float **d_c13, float **d_c23, float **d_Sw000, float **d_Sw001, float **d_Sw010, float **d_Sw011, float **d_Sw100, float **d_Sw101, float **d_Sw110, float **d_Sw111, int **d_Sjz, int **d_Sjx, int **d_Sjy, float **d_Rw000, float **d_Rw001, float **d_Rw010, float **d_Rw011, float **d_Rw100, float **d_Rw101, float **d_Rw110, float **d_Rw111, int **d_Rjz, int **d_Rjx, int **d_Rjy, float **d_bell, float **d_ww, float **d_ro, float **d_bzl_s, float **d_bzh_s, float **d_bxl_s, float **d_bxh_s, float **d_byl_s, float **d_byh_s, float *** uoz, float *** uox, float *** uoy, float *h_uoz, float *h_uox, float *h_uoy, float ***uc, float *h_dd, float *h_dd_combined, float **d_dd, sf_axis az, sf_axis ax, sf_axis ay, const int *nylocal, float spo, float idx, float idy, float idz, int nt, int jsnap, int jdata, int ngpu, int nyinterior, int ns, int nr, int nbell, int nc, bool interp, bool snap, bool fsrf, bool ssou, bool dabc, bool verb)
{
  int nb = fdm->nb;
  int nx = fdm->nx;

  if(verb) fprintf(stderr,"\n");
  for (int it=0; it<nt; it++) {
    if(verb) fprintf(stderr,"\b\b\b\b\b%d",it);

    /*------------------------------------------------------------*/
    /* from displacement to strain                                */
    /*    - Compute strains from displacements as in equation 1 */
    /*      - Step #1 (Steps denoted are as in Figure 2)    */
    /*------------------------------------------------------------*/
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      dim3 dimGrid2((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
      dim3 dimBlock2(24,24,1);
      dispToStrain<<<dimGrid2, dimBlock2, 32*32*3*sizeof(float)>>>(fdm->nxpad, nylocal[g], fdm->nzpad, d_uox[g], d_uoy[g], d_uoz[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g], idx, idy, idz);
    }
    sf_check_gpu_error("dispToStrain Kernel");

    /*------------------------------------------------------------*/
    /* from strain to stress                                      */
    /*    - Compute stress from strain as in equation 2     */
    /*      - Step #2                     */
    /*------------------------------------------------------------*/
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      dim3 dimGrid3(ceil(fdm->nxpad/192.0f), fdm->nzpad, nyinterior);
      dim3 dimBlock3(192,1,1);
      strainToStress<<<dimGrid3, dimBlock3>>>(g, fdm->nxpad, fdm->nzpad, nyinterior, d_c11[g], d_c12[g], d_c13[g], d_c22[g], d_c23[g], d_c33[g], d_c44[g], d_c55[g], d_c66[g], d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tyz[g], d_tzx[g]);
    }
    sf_check_gpu_error("strainToStress Kernel");


    /*------------------------------------------------------------*/
    /* free surface                                               */
    /*    - sets the z-component of stress tensor along the   */
    /*      free surface boundary to 0              */
    /*      - Step #3                     */
    /*------------------------------------------------------------*/
    if(fsrf) {
      for (int g = 0; g < ngpu; g++){
        hipSetDevice(g);
        dim3 dimGrid4(ceil(fdm->nxpad/8.0f), ceil(fdm->nb/8.0f), ceil(nyinterior/8.0f));
        dim3 dimBlock4(8,8,8);
        freeSurf<<<dimGrid4, dimBlock4>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, fdm->nb, d_tzz[g], d_tyz[g], d_tzx[g]);
      }
      sf_check_gpu_error("freeSurf Kernel");
    }


    /*------------------------------------------------------------*/
    /* inject stress source                                       */
    /*    - Step #4                       */
    /*------------------------------------------------------------*/
    if(ssou) {
      for (int g = 0; g < ngpu; g++){
        hipSetDevice(g);
        dim3 dimGrid5(ns, 1, 1);
        dim3 dimBlock5(2 * nbell + 1, 2 * nbell + 1, 1);
        lint3d_bell_gpu<<<dimGrid5, dimBlock5>>>(g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tzz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
        lint3d_bell_gpu<<<dimGrid5, dimBlock5>>>(g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_txx[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
        lint3d_bell_gpu<<<dimGrid5, dimBlock5>>>(g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_tyy[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
      }
      sf_check_gpu_error("lint3d_bell_gpu Kernel");
    }


    /*------------------------------------------------------------*/
    /* exchange halo regions of d_t arrays between GPUs           */
    /*------------------------------------------------------------*/
    if (ngpu > 1){ // using multiple GPUs, must exchange halo regions between neighboring GPUs

      // high halo region of d_t arrays on GPU 0 to GPU 1
      hipMemcpy(d_tzz[1], d_tzz[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tyy[1], d_tyy[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_txx[1], d_txx[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_txy[1], d_txy[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tyz[1], d_tyz[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tzx[1], d_tzx[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

      // exchange halo regions of d_t arrays between all internal GPUs
      for (int g = 1; g < ngpu-1; g++){
        // high halo region of GPU g to low halo region of GPU g+1
        hipMemcpy(d_tzz[g+1], d_tzz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tyy[g+1], d_tyy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_txx[g+1], d_txx[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_txy[g+1], d_txy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tyz[g+1], d_tyz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tzx[g+1], d_tzx[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

        // low halo region of GPU g to high halo region of GPU g-1
        hipMemcpy(d_tzz[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_tzz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tyy[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_tyy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_txx[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_txx[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_txy[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_txy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tyz[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_tyz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_tzx[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_tzx[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      }

      // low halo region of d_t arrays on GPU (ngpu-1) to GPU (ngpu-2)
      hipMemcpy(d_tzz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_tzz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tyy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_tyy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_txx[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_txx[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_txy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_txy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tyz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_tyz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_tzx[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_tzx[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

    }


    /*------------------------------------------------------------*/
    /* from stress to acceleration  (first term in RHS of eq. 3)  */
    /*    - Step #5                       */
    /*------------------------------------------------------------*/
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      dim3 dimGrid6((fdm->nxpad-2*NOP)/24.0f, (fdm->nzpad-2*NOP)/24.0f);
      dim3 dimBlock6(24,24,1);
      stressToAccel<<<dimGrid6, dimBlock6, 32*32*5*sizeof(float)>>>(fdm->nxpad, fdm->nzpad, nylocal[g], idx, idy, idz, d_txx[g], d_tyy[g], d_tzz[g], d_txy[g], d_tzx[g], d_tyz[g], d_uax[g], d_uay[g], d_uaz[g]);
    }
    sf_check_gpu_error("stressToAccel Kernel");


    /*------------------------------------------------------------*/
    /* inject acceleration source  (second term in RHS of eq. 3)  */
    /*    - Step #6                       */
    /*------------------------------------------------------------*/
    if(!ssou) {
      for (int g = 0; g < ngpu; g++){
        hipSetDevice(g);
        dim3 dimGrid7(ns, 1, 1);
        dim3 dimBlock7(2 * nbell + 1, 2 * nbell + 1, 1);
        lint3d_bell_gpu<<<dimGrid7, dimBlock7>>>(g, it, nc, ns, 0, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uaz[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
        lint3d_bell_gpu<<<dimGrid7, dimBlock7>>>(g, it, nc, ns, 1, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uax[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
        lint3d_bell_gpu<<<dimGrid7, dimBlock7>>>(g, it, nc, ns, 2, nbell, fdm->nxpad, nyinterior, fdm->nzpad, d_uay[g], d_bell[g], d_Sjx[g], d_Sjz[g], d_Sjy[g], d_ww[g], d_Sw000[g], d_Sw001[g], d_Sw010[g], d_Sw011[g], d_Sw100[g], d_Sw101[g], d_Sw110[g], d_Sw111[g]);
      }

      sf_check_gpu_error("lint3d_bell_gpu Kernel");
    }


    /*------------------------------------------------------------*/
    /* step forward in time                                       */
    /*    - Compute forward time step based on acceleration   */
    /*      - Step #7                     */
    /*------------------------------------------------------------*/
    for (int g = 0; g < ngpu; g++){
      hipSetDevice(g);
      dim3 dimGrid8(ceil(fdm->nxpad/192.0f), fdm->nzpad, nyinterior);
      dim3 dimBlock8(192,1,1);
      stepTime<<<dimGrid8, dimBlock8>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_ro[g], d_uox[g], d_umx[g], d_uax[g], d_upx[g], d_uoy[g], d_umy[g], d_uay[g], d_upy[g], d_uoz[g], d_umz[g], d_uaz[g], d_upz[g]);
    }
    sf_check_gpu_error("stepTime Kernel");


    /* circulate wavefield arrays */
    for (int g = 0; g < ngpu; g++){
      d_utz[g]=d_umz[g]; d_uty[g]=d_umy[g]; d_utx[g]=d_umx[g];
      d_umz[g]=d_uoz[g]; d_umy[g]=d_uoy[g]; d_umx[g]=d_uox[g];
      d_uoz[g]=d_upz[g]; d_uoy[g]=d_upy[g]; d_uox[g]=d_upx[g];
      d_upz[g]=d_utz[g]; d_upy[g]=d_uty[g]; d_upx[g]=d_utx[g];
    }


    /*------------------------------------------------------------*/
    /* apply boundary conditions                                  */
    /*    - Step #8                       */
    /*------------------------------------------------------------*/
    if(dabc) {
      /*---------------------------------------------------------------*/
      /* apply One-way Absorbing BC as in (Clayton and Enquist, 1977)  */
      /*---------------------------------------------------------------*/
      for (int g = 0; g < ngpu; g++){
        hipSetDevice(g);
        dim3 dimGrid_abc_XY(ceil(fdm->nxpad/32.0f),ceil(nyinterior/32.0f),2);
        dim3 dimBlock_abc_XY(32,32,1);
        abcone3d_apply_XY<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uox[g], d_umx[g], d_bzl_s[g], d_bzh_s[g]);
        abcone3d_apply_XY<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uoy[g], d_umy[g], d_bzl_s[g], d_bzh_s[g]);
        abcone3d_apply_XY<<<dimGrid_abc_XY,dimBlock_abc_XY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_umz[g], d_bzl_s[g], d_bzh_s[g]);

        dim3 dimGrid_abc_ZY(2, ceil(nyinterior/32.0f), ceil(fdm->nzpad/32.0f));
        dim3 dimBlock_abc_ZY(1,32,32);
        abcone3d_apply_ZY<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uox[g], d_umx[g], d_bxl_s[g], d_bxh_s[g]);
        abcone3d_apply_ZY<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uoy[g], d_umy[g], d_bxl_s[g], d_bxh_s[g]);
        abcone3d_apply_ZY<<<dimGrid_abc_ZY,dimBlock_abc_ZY>>>(g, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_umz[g], d_bxl_s[g], d_bxh_s[g]);
      }

      hipSetDevice(0);
      dim3 dimGrid_abc_XZ(ceil(fdm->nxpad/32.0f),1,ceil(fdm->nzpad/32.0f));
      dim3 dimBlock_abc_XZ(32,1,32);
      abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_uox[0], d_umx[0], d_byl_s[0]);
      abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_uoy[0], d_umy[0], d_byl_s[0]);
      abcone3d_apply_XZ_low<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, fdm->nzpad, d_uoz[0], d_umz[0], d_byl_s[0]);

      hipSetDevice(ngpu-1);
      abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, d_uox[ngpu-1], d_umx[ngpu-1], d_byh_s[ngpu-1]);
      abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, d_uoy[ngpu-1], d_umy[ngpu-1], d_byh_s[ngpu-1]);
      abcone3d_apply_XZ_high<<<dimGrid_abc_XZ,dimBlock_abc_XZ>>>(fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, d_uoz[ngpu-1], d_umz[ngpu-1], d_byh_s[ngpu-1]);


      /*---------------------------------------------------------------*/
      /* apply Sponge BC as in (Cerjan, et al., 1985)                  */
      /*---------------------------------------------------------------*/
      for (int g = 0; g < ngpu; g++){
        hipSetDevice(g);
        dim3 dimGrid_spng_XY(ceil(fdm->nxpad/192.0f),nyinterior,1);
        dim3 dimBlock_spng_XY(192,1,1);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);

        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);

        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);
        sponge3d_apply_XY<<<dimGrid_spng_XY,dimBlock_spng_XY>>>(g, d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb);


        dim3 dimGrid_spng_ZY(ceil(nb/8.0f),ceil(fdm->nzpad/8.0f),ceil(nyinterior/8.0f));
        dim3 dimBlock_spng_ZY(8,8,8);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_umz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_uoz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_upz[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);

        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_umx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_uox[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_upx[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);

        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_umy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_uoy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
        sponge3d_apply_ZY<<<dimGrid_spng_ZY,dimBlock_spng_ZY>>>(g, d_upy[g], fdm->nxpad, nyinterior, fdm->nzpad, nb, nx, spo);
      }


      hipSetDevice(0);
      dim3 dimGrid_spng_XZ(ceil(fdm->nxpad/192.0f),1,fdm->nzpad);
      dim3 dimBlock_spng_XZ(192,1,1);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umz[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoz[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upz[0], fdm->nxpad, fdm->nzpad, nb);

      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umx[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uox[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upx[0], fdm->nxpad, fdm->nzpad, nb);

      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umy[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoy[0], fdm->nxpad, fdm->nzpad, nb);
      sponge3d_apply_XZ_low<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upy[0], fdm->nxpad, fdm->nzpad, nb);

      hipSetDevice(ngpu-1);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umz[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoz[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upz[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);

      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umx[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uox[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upx[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);

      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_umy[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_uoy[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);
      sponge3d_apply_XZ_high<<<dimGrid_spng_XZ,dimBlock_spng_XZ>>>(d_upy[ngpu-1], fdm->nxpad, nylocal[ngpu-1], fdm->nzpad, nb);

      sf_check_gpu_error("Boundary Condition Kernels");
    }


    /*------------------------------------------------------------*/
    /* exchange halo regions of d_uo arrays between GPUs          */
    /*------------------------------------------------------------*/
    if (ngpu > 1){

      // high halo region of d_uo arrays on GPU 0 to GPU 1
      hipMemcpy(d_uox[1], d_uox[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_uoy[1], d_uoy[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_uoz[1], d_uoz[0] + (fdm->nxpad * fdm->nzpad * (nyinterior - 4)), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

      // exchange halo regions of d_uo arrays between all internal GPUs
      for (int g = 1; g < ngpu-1; g++){
        // high halo region of d_uo arrays on GPU g to low halo region on GPU g+1
        hipMemcpy(d_uox[g+1], d_uox[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_uoy[g+1], d_uoy[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_uoz[g+1], d_uoz[g] + (fdm->nxpad * fdm->nzpad * nyinterior), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

        // low halo region of d_uo arrays on GPU g to high halo region on GPU g-1
        hipMemcpy(d_uox[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_uox[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_uoy[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_uoy[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_uoz[g-1] + (fdm->nxpad * fdm->nzpad * (nylocal[g-1] - 4)), d_uoz[g] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      }

      // low halo region of d_uo arrays on GPU (ngpu-1) to GPU (ngpu-2)
      hipMemcpy(d_uox[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_uox[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_uoy[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_uoy[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(d_uoz[ngpu-2] + (fdm->nxpad * fdm->nzpad * (nylocal[ngpu-2] - 4)), d_uoz[ngpu-1] + (4 * fdm->nxpad * fdm->nzpad), 4 * fdm->nxpad * fdm->nzpad * sizeof(float), hipMemcpyDefault);

    }


    /*------------------------------------------------------------*/
    /* cut wavefield and save                     */
    /*    - Step #9                       */
    /*------------------------------------------------------------*/
    if(snap && it%jsnap==0) {

      // write GPU 0's portion of the wavefield into output arrays
      hipMemcpy(h_uox, d_uox[0], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(h_uoy, d_uoy[0], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
      hipMemcpy(h_uoz, d_uoz[0], nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
      for (int y = 0; y < nyinterior; y++){
        for (int z = 0; z < fdm->nzpad; z++){
          for (int x = 0; x < fdm->nxpad; x++){
            uox[y][x][z] = h_uox[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
            uoy[y][x][z] = h_uoy[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
            uoz[y][x][z] = h_uoz[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
          }
        }
      }


      // write other GPU's portions of wavefield data into output arrays
      for (int g = 1; g < ngpu; g++){
        hipMemcpy(h_uox, d_uox[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(h_uoy, d_uoy[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);
        hipMemcpy(h_uoz, d_uoz[g] + 4 * fdm->nzpad * fdm->nxpad, nyinterior * fdm->nzpad * fdm->nxpad * sizeof(float), hipMemcpyDefault);

        for (int y = 0; y < nyinterior; y++){
          for (int z = 0; z < fdm->nzpad; z++){
            for (int x = 0; x < fdm->nxpad; x++){
              uox[g * nyinterior + y][x][z] = h_uox[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
              uoy[g * nyinterior + y][x][z] = h_uoy[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
              uoz[g * nyinterior + y][x][z] = h_uoz[y * fdm->nzpad * fdm->nxpad + z * fdm->nxpad + x];
            }
          }
        }
      }

      // Write wavefield arrays to output file
      cut3d(uoz,uc,fdm,az,ax,ay);
      sf_floatwrite(uc[0][0],sf_n(ax)*sf_n(ay)*sf_n(az),Fwfl);

      cut3d(uox,uc,fdm,az,ax,ay);
      sf_floatwrite(uc[0][0],sf_n(ax)*sf_n(ay)*sf_n(az),Fwfl);

      cut3d(uoy,uc,fdm,az,ax,ay);
      sf_floatwrite(uc[0][0],sf_n(ax)*sf_n(ay)*sf_n(az),Fwfl);

    }

    /*------------------------------------------------------------*/
    /* extract receiver data                    */
    /*------------------------------------------------------------*/
    if(it%jdata==0) {
      if (interp){  // use interpolation
        for (int g = 0; g < ngpu; g++){
          hipSetDevice(g);
          hipMemset(d_dd[g], 0, nr*nc*sizeof(float));
          dim3 dimGrid_extract(MIN(nr,ceil(nr/1024.0f)), 1, 1);
          dim3 dimBlock_extract(MIN(nr, 1024), 1, 1);
          lint3d_extract_gpu<<<dimGrid_extract, dimBlock_extract>>>(g, d_dd[g], nr, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_uox[g], d_uoy[g], d_Rjz[g], d_Rjx[g], d_Rjy[g], d_Rw000[g], d_Rw001[g], d_Rw010[g], d_Rw011[g], d_Rw100[g], d_Rw101[g], d_Rw110[g], d_Rw111[g]);
        }
        sf_check_gpu_error("lint3d_extract kernel");
      }
      else {    // no interpolation
        for (int g = 0; g < ngpu; g++){
          hipSetDevice(g);
          hipMemset(d_dd[g], 0, nr*nc*sizeof(float));
          dim3 dimGrid_extract(MIN(nr,ceil(nr/1024.0f)), 1, 1);
          dim3 dimBlock_extract(MIN(nr, 1024), 1, 1);
          extract_gpu<<<dimGrid_extract, dimBlock_extract>>>(g, d_dd[g], nr, fdm->nxpad, nyinterior, fdm->nzpad, d_uoz[g], d_uox[g], d_uoy[g], d_Rjz[g], d_Rjx[g], d_Rjy[g]);
        }
        sf_check_gpu_error("extract_gpu kernel");

      }

      // copy GPU 0's receiver data into h_dd_combined
      hipMemcpy(h_dd_combined, d_dd[0], nr * nc * sizeof(float), hipMemcpyDefault);

      // add all other GPU's recever data to h_dd_combined
      for (int g = 1; g < ngpu; g++){
        hipMemcpy(h_dd, d_dd[g], nr * nc * sizeof(float), hipMemcpyDefault);
        for (int i = 0; i < nr * nc; i++){
          h_dd_combined[i] += h_dd[i];
        }
      }

      // write receiver data to output file
      sf_floatwrite(h_dd_combined, nr*nc, Fdat);
    }


  } // END MAIN LOOP

}
static void check_zx_dim(const fdm3d &fdm, int ngpu)
{
  // check that dimmeionsons are ok for FD kernels
  if ((fdm->nzpad - 8) % 24 != 0){
    sf_error("nz + 2*nb - 8 is not a multiple of 24");
  }
  if ((fdm->nxpad - 8) % 24 != 0){
    sf_error("nx + 2*nb - 8 is not a multiple of 24");
  }
  if ((fdm->nypad % ngpu) != 0){
    sf_error("You are using %d GPUs.\n(ny + 2*nb) must me a multiple of %d\nChange model dimensions or select a different number of GPUs", ngpu, ngpu);
  }
}
static void set_nylocal(const fdm3d &fdm, int *nylocal, int ngpu, int nyinterior)
{
  // all interior nodes need 8 additional ghost slices (4 on each side of the y axis)
  for (int g = 0; g < ngpu; g++){
    nylocal[g] = nyinterior + 8;
  }

  // exterior nodes only require 4 additional ghost slices
  if (ngpu >= 2){
    nylocal[0] = nyinterior + 4;
    nylocal[ngpu-1] = nyinterior + 4;
  }

  // if using 1 GPU, this GPU holds the entire domain
  if (ngpu == 1){
    nylocal[0] = fdm->nypad;
  }
}

static void make_axis(modeling_t *m, sf_axis &az, sf_axis &ax, sf_axis &ay)
{
  az = sf_maxa(m->n1 - 2 * m->nb, m->o1 + m->nb * m->d1, m->d1);
  ax = sf_maxa(m->n2 - 2 * m->nb, m->o2 + m->nb * m->d2, m->d2);
  ay = sf_maxa(m->n3 - 2 * m->nb, m->o3 + m->nb * m->d3, m->d3);
}

static void run(sf_file Fwfl, sf_file Fdat, fdm3d &oldfdm, pt3d *ss, pt3d *rr, sf_axis az, sf_axis ax, sf_axis ay, int nt, float dt, const float *h_ro, const float *h_c11, const float *h_c22, const float *h_c33, const float *h_c44, const float *h_c55, const float *h_c66, const float *h_c12, const float *h_c13, const float *h_c23, float **d_ww, int ns, int nr, int nb, int ngpu, int jdata, int jsnap, int nbell, int nc, bool interp, bool ssou,  bool dabc, bool snap, bool fsrf, bool verb)
{

  /*------------------------------------------------------------*/
  // used for writing wavefield to file, only needed if snap=y
  float ***uox, ***uoy, ***uoz;
  float *h_uox, *h_uoy, *h_uoz;
  uox = uoy = uoz = NULL;
  h_uox = h_uoy = h_uoz = NULL;

  /* wavefield cut params */
  float     ***uc=NULL;

  float   idz,idx,idy;

  float *h_dd, *h_dd_combined, **d_dd;
  setup_output_array(h_dd, h_dd_combined, d_dd, ngpu, nr, nc);

  float **d_bell = setup_bell(nbell, ngpu);
  /*------------------------------------------------------------*/

  // TODO: put time block stuff here
  /* expand domain for FD operators and ABC */
  fdm3d fdm=fdutil3d_init(verb,fsrf,az,ax,ay,nb,1);
  update_axis(fdm, az, ax, ay, verb);

  /*------------------------------------------------------------*/
  /* compute sub-domain dimmensions (domain decomposition) */
  int nyinterior = (fdm->nypad / ngpu);   // size of sub-domains in y-dimension EXCLUDING any ghost cells from adjacent GPUs
  int *nylocal = (int*)malloc(ngpu*sizeof(int));  // size of sub-domains in y-dimension INCLUDING any ghost cells from adjacent GPUs
  set_nylocal(fdm, nylocal, ngpu, nyinterior);
  check_zx_dim(fdm, ngpu);
  if(snap) { alloc_wlf(fdm, uoz, uox, uoy, h_uoz, h_uox, h_uoy, uc, nyinterior); }
  float **d_Sw000,  **d_Sw001,  **d_Sw010,  **d_Sw011,  **d_Sw100,  **d_Sw101,  **d_Sw110,  **d_Sw111;
  float **d_Rw000,  **d_Rw001,  **d_Rw010,  **d_Rw011,  **d_Rw100,  **d_Rw101,  **d_Rw110,  **d_Rw111;
  int **d_Sjz,  **d_Sjx,  **d_Sjy;
  int **d_Rjz,  **d_Rjx,  **d_Rjy;

  /* calculate 3d linear interpolation coefficients for source/receiver locations and copy to each GPU*/
  setup_interp_cooef(d_Sw000, d_Sw001, d_Sw010, d_Sw011, d_Sw100, d_Sw101, d_Sw110, d_Sw111, d_Sjz, d_Sjx, d_Sjy, fdm, ss, ns, ngpu);
  setup_interp_cooef(d_Rw000, d_Rw001, d_Rw010, d_Rw011, d_Rw100, d_Rw101, d_Rw110, d_Rw111, d_Rjz, d_Rjx, d_Rjy, fdm, rr, nr, ngpu);

  setup_fd_cooef(fdm, idz, idx, idy);
  float **d_ro ,  **d_c11,  **d_c22,  **d_c33,  **d_c44,  **d_c55,  **d_c66,  **d_c12,  **d_c13,  **d_c23;
  // TODO: we need interpolation here for h_ro, h_c11 ... h_c23
  // TODO: rename h_ro and other similar to full_h_ro
  copy_den_vel_to_dev(fdm, d_ro , d_c11, d_c22, d_c33, d_c44, d_c55, d_c66, d_c12, d_c13, d_c23, h_ro, h_c11, h_c22, h_c33, h_c44, h_c55, h_c66, h_c12, h_c13, h_c23, nyinterior, ngpu);

  float spo = 0;
  float **d_bzl_s,  **d_bzh_s,  **d_bxl_s,  **d_bxh_s,  **d_byl_s,  **d_byh_s;
  setup_boundary(fdm, d_bzl_s, d_bzh_s, d_bxl_s, d_bxh_s, d_byl_s, d_byh_s, h_ro, h_c55, spo, nyinterior, ngpu, dt, dabc);

  float **d_umx,  **d_uox,  **d_upx,  **d_uax,  **d_utx,  **d_umy,  **d_uoy,  **d_upy,  **d_uay,  **d_uty,  **d_umz,  **d_uoz,  **d_upz,  **d_uaz,  **d_utz,  **d_tzz,  **d_txx,  **d_tyy,  **d_txy,  **d_tyz,  **d_tzx;
  init_wfd_array(fdm, d_umx, d_uox, d_upx, d_uax, d_utx, d_umy, d_uoy, d_upy, d_uay, d_uty, d_umz, d_uoz, d_upz, d_uaz, d_utz, d_tzz, d_txx, d_tyy, d_txy, d_tyz, d_tzx, nylocal, ngpu);

  precompute(fdm, d_ro, dt, nyinterior, ngpu);


  /*------------------------------------------------------------*/
  /*
   *  MAIN LOOP
   */
  /*------------------------------------------------------------*/
  main_loop(Fwfl, Fdat, fdm, d_umx, d_uox, d_upx, d_uax, d_utx, d_umy, d_uoy, d_upy, d_uay, d_uty, d_umz, d_uoz, d_upz, d_uaz, d_utz, d_tzz, d_txx, d_tyy, d_txy, d_tyz, d_tzx, d_c11, d_c22, d_c33, d_c44, d_c55, d_c66, d_c12, d_c13, d_c23, d_Sw000, d_Sw001, d_Sw010, d_Sw011, d_Sw100, d_Sw101, d_Sw110, d_Sw111, d_Sjz, d_Sjx, d_Sjy, d_Rw000, d_Rw001, d_Rw010, d_Rw011, d_Rw100, d_Rw101, d_Rw110, d_Rw111, d_Rjz, d_Rjx, d_Rjy, d_bell, d_ww, d_ro, d_bzl_s, d_bzh_s, d_bxl_s, d_bxh_s, d_byl_s, d_byh_s,  uoz,  uox,  uoy, h_uoz, h_uox, h_uoy, uc, h_dd, h_dd_combined, d_dd, az, ax, ay, nylocal, spo, idx, idy, idz, nt, jsnap, jdata, ngpu, nyinterior, ns, nr, nbell, nc, interp, snap, fsrf, ssou, dabc, verb);

  /*------------------------------------------------------------*/
  /* deallocate host arrays */

  free(h_dd); free(h_dd_combined);
  //free(ss); free(rr);
  //free(h_ro);
  //free(h_c11); free(h_c22); free(h_c33); free(h_c44); free(h_c55); free(h_c66); free(h_c12); free(h_c13); free(h_c23);

  if (snap){
    free(h_uoz); free(h_uox); free(h_uoy);
    free(**uc);  free(*uc);  free(uc);
    free(**uoz); free(*uoz); free(uoz);
    free(**uox); free(*uox); free(uox);
    free(**uoy); free(*uoy); free(uoy);
  }


  /*------------------------------------------------------------*/
  /* deallocate GPU arrays */

  for (int g = 0; g < ngpu; g++){

    //hipFree(&d_ww[g]);
    hipFree(&d_dd[g]);
    hipFree(&d_bell[g]);

    hipFree(&d_ro[g]);
    hipFree(&d_c11[g]);
    hipFree(&d_c22[g]);
    hipFree(&d_c33[g]);
    hipFree(&d_c44[g]);
    hipFree(&d_c55[g]);
    hipFree(&d_c66[g]);
    hipFree(&d_c12[g]);
    hipFree(&d_c13[g]);
    hipFree(&d_c23[g]);

    if (dabc){
      hipFree(&d_bzl_s[g]);
      hipFree(&d_bzh_s[g]);
      hipFree(&d_bxl_s[g]);
      hipFree(&d_bxh_s[g]);
      hipFree(&d_byl_s[0]);
      hipFree(&d_byh_s[ngpu-1]);
    }

    hipFree(&d_umx[g]); hipFree(&d_umy[g]); hipFree(&d_umz[g]);
    hipFree(&d_uox[g]); hipFree(&d_uoy[g]); hipFree(&d_uoz[g]);
    hipFree(&d_upx[g]); hipFree(&d_upy[g]); hipFree(&d_upz[g]);
    hipFree(&d_uax[g]); hipFree(&d_uay[g]); hipFree(&d_uaz[g]);

    hipFree(&d_tzz[g]); hipFree(&d_tyy[g]); hipFree(&d_txx[g]);
    hipFree(&d_txy[g]); hipFree(&d_tyz[g]); hipFree(&d_tzx[g]);

    hipFree(&d_Sjz[g]);
    hipFree(&d_Sjx[g]);
    hipFree(&d_Sjy[g]);
    hipFree(&d_Sw000[g]);
    hipFree(&d_Sw001[g]);
    hipFree(&d_Sw010[g]);
    hipFree(&d_Sw011[g]);
    hipFree(&d_Sw100[g]);
    hipFree(&d_Sw101[g]);
    hipFree(&d_Sw110[g]);
    hipFree(&d_Sw111[g]);

    hipFree(&d_Rjz[g]);
    hipFree(&d_Rjx[g]);
    hipFree(&d_Rjy[g]);
    if (interp){
      hipFree(&d_Rw000[g]);
      hipFree(&d_Rw001[g]);
      hipFree(&d_Rw010[g]);
      hipFree(&d_Rw011[g]);
      hipFree(&d_Rw100[g]);
      hipFree(&d_Rw101[g]);
      hipFree(&d_Rw110[g]);
      hipFree(&d_Rw111[g]);
    }
  }
}

// entry point
int main(int argc, char* argv[]) {

  bool verb,fsrf,snap,ssou,dabc,interp;
  int  jsnap,jdata;

  /* I/O files */
  sf_file Fwav=NULL; /* wavelet   */
  sf_file Fsou=NULL; /* sources   */
  sf_file Frec=NULL; /* receivers */
  sf_file Fccc=NULL; /* velocity  */
  sf_file Fden=NULL; /* density   */
  sf_file Fdat=NULL; /* data      */
  sf_file Fwfl=NULL; /* wavefield */

  /* cube axes */
  sf_axis at,ax,ay,az;
  sf_axis as,ar;

  int     nt,nz,nx,ny,ns,nr,nb;
  float   dt;

  /* I/O arrays */
  pt3d   *ss=NULL;           /* sources   */
  pt3d   *rr=NULL;           /* receivers */


  /* Gaussian bell */
  int nbell;


  /* init RSF */
  sf_init(argc,argv);


  /*------------------------------------------------------------*/
  /* init GPU */
  int ngpu;
  if (! sf_getint("ngpu", &ngpu)) ngpu = 1; /* how many local GPUs to use */
  sf_warning("using %d GPUs", ngpu);
  for (int g = 0; g < ngpu; g++){
    hipSetDevice(g);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  }


  /*------------------------------------------------------------*/
  /* execution flags */
  if(! sf_getbool("verb",&verb)) verb=false; /* verbosity flag */
  if(! sf_getbool("snap",&snap)) snap=false; /* wavefield snapshots flag */
  if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
  if(! sf_getbool("ssou",&ssou)) ssou=false; /* stress source */
  if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
  if(! sf_getbool("interp",&interp)) interp=true; /* perform linear interpolation on receiver data */


  /*------------------------------------------------------------*/
  /* I/O files */
  Fwav = sf_input ("in"); /* wavelet   */
  Fccc = sf_input ("ccc"); /* stiffness */
  Fden = sf_input ("den"); /* density   */
  Fsou = sf_input ("sou"); /* sources   */
  Frec = sf_input ("rec"); /* receivers */
  Fwfl = sf_output("wfl"); /* wavefield */
  Fdat = sf_output("out"); /* data      */


  /*------------------------------------------------------------*/
  /* axes */
  at = sf_iaxa(Fwav,3); sf_setlabel(at,"t"); if(verb) sf_raxa(at); /* time */
  az = sf_iaxa(Fccc,1); sf_setlabel(az,"z"); if(verb) sf_raxa(az); /* depth */
  ax = sf_iaxa(Fccc,2); sf_setlabel(ax,"x"); if(verb) sf_raxa(ax); /* space x */
  ay = sf_iaxa(Fccc,3); sf_setlabel(ay,"y"); if(verb) sf_raxa(ay); /* space y */

  sf_axis asz, asx, asy;
  asz = sf_iaxa(Fsou, 2); asx = sf_iaxa(Fsou, 3); asy = sf_iaxa(Fsou, 4);
  as = sf_maxa(sf_n(asz) * sf_n(asx) * sf_n(asy), sf_d(asx), sf_o(asx));

  sf_axis arz, arx, ary;
  arz = sf_iaxa(Frec, 2); arx = sf_iaxa(Frec, 3); ary = sf_iaxa(Frec, 4);
  ar = sf_maxa(sf_n(arz) * sf_n(arx) * sf_n(ary), sf_d(arx), sf_o(arx));

  nt = sf_n(at); dt = sf_d(at);
  nz = sf_n(az);
  nx = sf_n(ax);
  ny = sf_n(ay);

  ns = sf_n(as);
  nr = sf_n(ar);


  /*------------------------------------------------------------*/
  /* other execution parameters */
  if(! sf_getint("nbell",&nbell)) nbell=5;  /* bell size */
  if(verb) sf_warning("nbell=%d",nbell);
  if(! sf_getint("jdata",&jdata)) jdata=1;  /* extract receiver data every jdata time steps */
  if(snap) {
    if(! sf_getint("jsnap",&jsnap)) jsnap=nt;  /* save wavefield every jsnap time steps */
  }

  if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;

  /*------------------------------------------------------------*/
  /* 3D vector components */
  int nc=3;
  sf_axis ac=sf_maxa(nc  ,0,1);
  setup_output_data(Fdat, at, ar, ac, nt, jdata, dt);

  float **d_ww = init_wavelet(Fwav, ns, nc, nt, ngpu);

  sf_axis full_az = sf_maxa(sf_n(az), sf_o(az), sf_d(az));
  sf_axis full_ax = sf_maxa(sf_n(ax), sf_o(ax), sf_d(ax));
  sf_axis full_ay = sf_maxa(sf_n(ay), sf_o(ay), sf_d(ay));
  fdm3d fullfdm=fdutil3d_init(verb,fsrf,full_az,full_ax,full_ay,nb,1);
  update_axis(fullfdm, full_az, full_ax, full_ay, verb);
  if (snap)  set_output_wfd(Fwfl, at, full_az, full_ax, full_ay, ac, nt, dt, jsnap, verb);

  setup_src_rcv_cord(Fsou, Frec, ss, rr, ns, nr);

  float ***full_h_ro,  ***full_h_c11,  ***full_h_c22,  ***full_h_c33,  ***full_h_c44,  ***full_h_c55,  ***full_h_c66,  ***full_h_c12,  ***full_h_c13,  ***full_h_c23;
  read_density_velocity(Fden, Fccc, fullfdm, full_h_ro, full_h_c11, full_h_c22, full_h_c33, full_h_c44, full_h_c55, full_h_c66, full_h_c12, full_h_c13, full_h_c23, nz, nx, ny);

  sf_warning("begin conghui's code");
  int   timeblocks;
  float vmin;
  float vmax;
  float dmin;
  float dmax;
  float maxf; // maximum frequency
  float   error;
  float errorfact;
  float qfact;
  float downfact;
  float w0; // for velocity
  bool withq;

  if (!sf_getint("timeblocks", &timeblocks)) timeblocks = 40;
  if (!sf_getfloat("maxf", &maxf)) maxf = 80;
  if (!sf_getfloat("error", &error)) error = 20;
  if (!sf_getfloat("errorfact", &errorfact)) errorfact = 1.2;
  if (!sf_getfloat("downfact", &downfact)) downfact = 0.04;
  if (!sf_getfloat("qfact", &qfact)) qfact = 50; // copy from vel_mod.f90
  if (!sf_getfloat("w0", &w0)) w0 = 60;
  if (!sf_getbool("withq", &withq)) withq = false;

  sf_file Fvelp = sf_input("vp"); // p wave velocity
  float ***v0 = sf_floatalloc3(nz, nx, ny);
  sf_seek(Fvelp, 0, SEEK_SET);
  sf_floatread(v0[0][0], nx*ny*nz, Fvelp);
  vel_t *vv0 = clone_vel(v0, nz, nx, ny, sf_o(az), sf_o(ax), sf_o(ay), sf_d(az), sf_d(ax), sf_d(ay), w0, qfact);
  vmin_vmax_dmin_dmax(vv0, &vmin, &vmax, &dmin, &dmax);

  sf_warning("vmin: %f, vmax: %f, dmin: %f, dmax: %f", vmin, vmax, dmin, dmax);
  times_t *times = read_times();
  init_box(timeblocks, vmin, vmax, dmin, dmax, maxf, nb, error, errorfact, qfact, downfact);
  box_t *domain = calc_shot_box(vv0, times, ss, rr, nr, nt, dt);

  init_sinc_table(8, 10000);
  modeling_t initmodel = make_modeling(vv0);

  fdm3d oldfdm = clonefdm(fullfdm);

  // initialize host prev and current wavefield
  float ***h_umx, ***h_uox,  ***h_umy,  ***h_uoy,  ***h_umz,  ***h_uoz;
  init_host_umo(oldfdm, h_umx, h_uox,  h_umy,  h_uoy,  h_umz,  h_uoz);

  for (int iblock = 0; iblock < domain->timeblocks; iblock++) {
    sf_warning("FORWARD BLOCK: %d", iblock);

    sf_axis curaz = sf_maxa(1,1,1); // dummy, update later
    sf_axis curax = sf_maxa(1,1,1); // dummy, update later
    sf_axis curay = sf_maxa(1,1,1); // dummy, update later
    modeling_t *cur = &domain->hyper[iblock];

    make_axis(cur, curaz, curax, curay);
  }

  // TODO: put your code here, update az, ax, zy, nt, dt, then everything is supposed to be fine
  // TODO: you also need to interpolate full_*
  run(Fwfl, Fdat, oldfdm, ss, rr, az, ax, ay, nt, dt, full_h_ro[0][0], full_h_c11[0][0], full_h_c22[0][0], full_h_c33[0][0], full_h_c44[0][0], full_h_c55[0][0], full_h_c66[0][0], full_h_c12[0][0], full_h_c13[0][0], full_h_c23[0][0], d_ww, ns, nr, nb, ngpu, jdata, jsnap, nbell, nc, interp, ssou,  dabc, snap, fsrf, verb);

  /*------------------------------------------------------------*/
  /* deallocate host arrays */
  release_host_umo(h_umx, h_uox,  h_umy,  h_uoy,  h_umz,  h_uoz);
  free(ss); free(rr);
  free(**full_h_ro); free(*full_h_ro); free(full_h_ro);
  free(**full_h_c11); free(**full_h_c22); free(**full_h_c33); free(**full_h_c44); free(**full_h_c55); free(**full_h_c66); free(**full_h_c12); free(**full_h_c13); free(**full_h_c23);
  free(*full_h_c11); free(*full_h_c22); free(*full_h_c33); free(*full_h_c44); free(*full_h_c55); free(*full_h_c66); free(*full_h_c12); free(*full_h_c13); free(*full_h_c23);
  free(full_h_c11); free(full_h_c22); free(full_h_c33); free(full_h_c44); free(full_h_c55); free(full_h_c66); free(full_h_c12); free(full_h_c13); free(full_h_c23);

  /*------------------------------------------------------------*/
  /* deallocate GPU arrays */
  for (int g = 0; g < ngpu; g++){
    hipFree(&d_ww[g]);
  }

  sf_close();
  exit(0);

}



